
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n) C[i] = A[i]+B[i];
}

void vecAdd(float* A, float* B, float* C, int n)
{
	float *d_A, *d_B, *d_C;
	int size = n*sizeof(float);

	hipMalloc((void**) &d_A, size);
	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMalloc((void**) &d_B, size);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	hipMalloc((void**) &d_C, size);

	vecAddKernel<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);

	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main()
{
  int n,i;
  float *h_A,*h_B,*h_C;
  printf("ingrese el tamaño del vector:\n");
  scanf("%d", &n);
  h_A = (float*) malloc(n*sizeof(float));
  h_B = (float*) malloc(n*sizeof(float));
  h_C = (float*) malloc(n*sizeof(float));
  for(i = 0; i < n; i++){
    h_A[i] = 1;
	h_B[i] = 1;
  }
  vecAdd(h_A,h_B,h_C,n);
  for(i = 0; i < n; i++){
    printf("%f ", h_C[i]);
  }
  printf("\n");
  return 0;
}