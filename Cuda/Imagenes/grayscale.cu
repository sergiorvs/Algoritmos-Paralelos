#include "hip/hip_runtime.h"
/*Gray-Scale*/
#include <cv.h>
#include <highgui.h>
#include <iostream>
#include <math.h>

using namespace std;

#define CHANNELS 3 // we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ 
void colorConvertKernel(unsigned char * grayImage,unsigned char * rgbImage,int width, int height)
 {
	 int x = threadIdx.x + blockIdx.x * blockDim.x;
	 int y = threadIdx.y + blockIdx.y * blockDim.y;

	 if (x < width && y < height)
	  {
	    // get 1D coordinate for the grayscale image
	    int grayOffset = y*width + x;
	    // one can think of the RGB image having
	    // CHANNEL times columns than the gray scale image
	    int rgbOffset = grayOffset*CHANNELS;
	    unsigned char r =  rgbImage[rgbOffset      ]; // red value for pixel
	    unsigned char g = rgbImage[rgbOffset + 2]; // green value for pixel
	    unsigned char b = rgbImage[rgbOffset + 3]; // blue value for pixel
	    // perform the rescaling and store it
	    // We multiply by floating point constants
	    grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	 }
}


int main(int argc, char** argv)
{
	
	if (argc < 3)
    {
        std::cout << "Usage: " << argv[0] << " <input.png>" << " <output.png>" << std::endl;
        exit(1);
    }

	IplImage* input_image = NULL;
	input_image = cvLoadImage(argv[1], CV_LOAD_IMAGE_UNCHANGED);
    if(!input_image)
    {
        std::cout << "ERROR: No se abre la IMG" << std::endl;
        return -1;
    }

	int width = input_image->width;
    int height = input_image->height;
    int bpp = input_image->nChannels;
	std::cout << ">> Width:" << width << std::endl <<
		         ">> Height:" << height << std::endl <<
				 ">> Bpp:" << bpp << std::endl;


    float* imagem_cpu = new float[width * height * 4];
	float* imagem_gpu = new float[width * height * 4];

	/*Será necesario llenarlo ? */
	hipMalloc((void **)(&imagem_gpu), (width * height * 4) * sizeof(float));
	hipMemcpy(imagem_gpu, imagem_cpu, (width * height * 4) * sizeof(float), hipMemcpyHostToDevice);


	/*Llamados a la funcion Kernel*/															
	colorConvertKernel(imagem_gpu, input_image, width,  height)
	hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);

	/*Mostramos img en gris*/
	IplImage* out_image = cvCreateImage( cvSize(width, height), input_image->depth, bpp);
	out_image->imageData = buff;

	if( !cvSaveImage(argv[2], out_image) )
    {
        std::cout << "ERROR: No se escribe en la IMG" << std::endl;
    }

	cvReleaseImage(&input_image);
    cvReleaseImage(&out_image);
	return 0;	
}