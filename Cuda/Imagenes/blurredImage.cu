#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <iostream>
#include <math.h>

using namespace std;

#define CHANNELS 3 // we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ 
  void blurKernel(unsigned char * in, unsigned char * out, int w, int h) {
      int Col  = blockIdx.x * blockDim.x + threadIdx.x;
      int Row  = blockIdx.y * blockDim.y + threadIdx.y;

      if (Col < w && Row < h) {
          int pixVal = 0;
          int pixels = 0;

          // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
          for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
              for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {

                  int curRow = Row + blurRow;
                  int curCol = Col + blurCol;
                  // Verify we have a valid image pixel
                  if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                      pixVal += in[curRow * w + curCol];
                      pixels++; // Keep track of number of pixels in the accumulated total
                  }
              }
          }

          // Write our new pixel value out
          out[Row * w + Col] = (unsigned char)(pixVal / pixels);
      }
  }

int main(int argc, char** argv)
{
	
	if (argc < 3)
    {
        std::cout << "Usage: " << argv[0] << " <input.png>" << " <output.png>" << std::endl;
        exit(1);
    }

	IplImage* input_image = NULL;
	input_image = cvLoadImage(argv[1], CV_LOAD_IMAGE_UNCHANGED);
    if(!input_image)
    {
        std::cout << "ERROR: No se abre la IMG" << std::endl;
        return -1;
    }

	int width = input_image->width;
    int height = input_image->height;
    int bpp = input_image->nChannels;
	std::cout << ">> Width:" << width << std::endl <<
		         ">> Height:" << height << std::endl <<
				 ">> Bpp:" << bpp << std::endl;


    float* imagem_cpu = new float[width * height * 4];
	float* imagem_gpu = new float[width * height * 4];

	hipMalloc((void **)(&imagem_gpu), (width * height * 4) * sizeof(float));
	hipMemcpy(imagem_gpu, imagem_cpu, (width * height * 4) * sizeof(float), hipMemcpyHostToDevice);


	/*Llamados a la funcion Kernel*/															
	colorConvertKernel(imagem_gpu, input_image, width,  height)
	hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);

	/*Mostramos img en gris*/
	IplImage* out_image = cvCreateImage( cvSize(width, height), input_image->depth, bpp);
	out_image->imageData = buff;

	if( !cvSaveImage(argv[2], out_image) )
    {
        cout << "ERROR: No se escribe en la IMG" << endl;
    }

	cvReleaseImage(&input_image);
    cvReleaseImage(&out_image);
	return 0;	
}