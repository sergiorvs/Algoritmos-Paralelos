
//compilar: nvcc matVec.cu -o c


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void matAddKernel(float *A, float *B, float *C, int n){
  int i = threadIdx.x + blockDim.x * blockIdx.x, j;

  if(i < n){
    C[i] = 0;
    for(j = 0; j < n; j++){
       C[i] += A[i*n+j] * B[j];
    }
  }
  
}

void matAdd(float* A, float* B, float* C, int n){
  int size = n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size*n);
  hipMemcpy(d_A,A,size*n,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);
  //printf("%f\n", B[0]);
  matAddKernel<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i,j;
  float *h_A,*h_B,*h_C;
  printf("ingrese el tamaño de la matriz:\n");
  scanf("%d", &n);
  h_A = (float*) malloc(n*n*sizeof(float));
  h_B = (float*) malloc(n*sizeof(float));
  h_C = (float*) malloc(n*sizeof(float));
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++)
      h_A[i*n+j] = 1;
  }
  for(i = 0; i < n; i++){
      h_B[i] = 1;
      h_C[i] = 0;
  }
  matAdd(h_A,h_B,h_C,n);
  for(i = 0; i < n; i++){
    printf("%f ", h_C[i]);	
  }
  printf("\n");
  free(h_A); free(h_B); free(h_C);
  return 0;
}